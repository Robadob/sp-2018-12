#include "hip/hip_runtime.h"
/**
 *	Based off earlier start from:
 *	https://github.com/Robadob/SP-Bench/commit/35dcbb81cc0b73cdb6b08fb622f13e688a878133
 */
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <glm/glm.hpp>
#include <glm/gtx/component_wise.hpp>
#include <glm/gtc/constants.hpp>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hipcub/hipcub.hpp>
#include <glm/gtc/epsilon.hpp>
#define EPSILON 0.005f
//#define CIRCLES
//Cuda call
static void HandleCUDAError(const char *file,
	int line,
	hipError_t status = hipGetLastError()) {
#ifdef _DEBUG
	hipDeviceSynchronize();
#endif
	if (status != hipError_t::hipSuccess || (status = hipGetLastError()) != hipError_t::hipSuccess)
	{
		printf("%s(%i) CUDA Error Occurred;\n%s\n", file, line, hipGetErrorString(status));
#ifdef _DEBUG
		getchar();
#endif
		exit(1);
	}
}
#define CUDA_CALL( err ) (HandleCUDAError(__FILE__, __LINE__ , err))
#define CUDA_CHECK() (HandleCUDAError(__FILE__, __LINE__))

//Logging (found in log.cpp)
#include <fstream>
void createLog(std::ofstream &f);
void log(std::ofstream &f,
	const unsigned int &estRadialNeighbours,
	const unsigned int &agentCount,
	const unsigned int &envWidth,
	const float &PBM_control,
	const float &kernel_control,
	const float &PBM,
	const float &kernel,
	const unsigned int &fails
);
__device__ __constant__ unsigned int d_agentCount;
__device__ __constant__ float d_environmentWidth_float;
__device__ __constant__ unsigned int d_gridDim;
glm::uvec2 GRID_DIMS;
__device__ __constant__ float d_gridDim_float;
__device__ __constant__ float d_RADIUS;
__device__ __constant__ float d_R_SIN_45;
__device__ __constant__ float d_binWidth;

__device__ __constant__ unsigned int d_SHARED_MESSAGE_COUNT;
unsigned int SHARED_MESSAGE_COUNT;

//For thread block max bin check
unsigned int *d_PBM_max_count;
unsigned int PBM_max_count = 0;
unsigned int PBM_max_Moore_count = 0;

texture<float2> d_texMessages;
texture<unsigned int> d_texPBM;

__global__ void init_curand(hiprandState *state, unsigned long long seed) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < d_agentCount)
		hiprand_init(seed, id, 0, &state[id]);
}
__global__ void init_agents(hiprandState *state, glm::vec2 *locationMessages) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= d_agentCount)
		return;
	//curand_unform returns 0<x<=1.0, not much can really do about 0 exclusive
	//negate and  + 1.0, to make  0<=x<1.0
	locationMessages[id].x = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentWidth_float;
	locationMessages[id].y = (-hiprand_uniform(&state[id]) + 1.0f)*d_environmentWidth_float;
}
__device__ __forceinline__ glm::ivec2 getGridPosition(glm::vec2 worldPos)
{
	//Clamp each grid coord to 0<=x<dim
	return clamp(floor((worldPos / d_environmentWidth_float)*d_gridDim_float), glm::vec2(0), glm::vec2((float)d_gridDim - 1));
}
__device__ __forceinline__ unsigned int getHash(glm::ivec2 gridPos)
{
	//Bound gridPos to gridDimensions
	gridPos = clamp(gridPos, glm::ivec2(0), glm::ivec2(d_gridDim - 1));
	//Compute hash (effectivley an index for to a bin within the partitioning grid in this case)
	return (unsigned int)(
		(gridPos.y * d_gridDim) +					//y
		gridPos.x); 	                            //x
}
__global__ void atomicHistogram(unsigned int* bin_index, unsigned int* bin_sub_index, unsigned int *pbm_counts, glm::vec2 *messageBuffer)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	glm::ivec2 gridPos = getGridPosition(messageBuffer[index]);
	unsigned int hash = getHash(gridPos);
	bin_index[index] = hash;
	unsigned int bin_idx = atomicInc((unsigned int*)&pbm_counts[hash], 0xFFFFFFFF);
	bin_sub_index[index] = bin_idx;
}
__global__ void reorderLocationMessages(
	unsigned int* bin_index,
	unsigned int* bin_sub_index,
	unsigned int *pbm,
	glm::vec2 *unordered_messages,
	glm::vec2 *ordered_messages
)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	unsigned int i = bin_index[index];
	unsigned int sorted_index = pbm[i] + bin_sub_index[index];

	//Order messages into swap space
	ordered_messages[sorted_index] = unordered_messages[index];
}
int requiredSM(int blockSize)
{
	return (SHARED_MESSAGE_COUNT*sizeof(float2))+(6*sizeof(unsigned int));//Need to limit this to the max SM
}
/**
* Kernel must be launched 1 block per bin
* This removes the necessity of __launch_bounds__(64) as all threads in block are touching the same messages
* However we end up with alot of (mostly) idle threads if one bin dense, others empty.
*/
__global__  void __launch_bounds__(64) neighbourSearch_control(const glm::vec2 *agents, glm::vec2 *out)
{
#define STRIPS
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;
	glm::vec2 pos = agents[index];
	glm::ivec2 gridPos = getGridPosition(pos);
	glm::ivec2 gridPosRelative;
	unsigned int count = 0;
	glm::vec2 average = glm::vec2(0);

	//if (index == 9)
	//	printf("(%d, %d)\n", gridPos.x, gridPos.y);
	for (gridPosRelative.y = -1; gridPosRelative.y <= 1; gridPosRelative.y++)
	{//ymin to ymax
		int currentBinY = gridPos.y + gridPosRelative.y;
		if (currentBinY >= 0 && currentBinY < d_gridDim)
		{
#ifndef STRIPS
			for (gridPosRelative.x = -1; gridPosRelative.x <= 1; gridPosRelative.x++)
			{//xmin to xmax
				int currentBinX = gridPos.x + gridPosRelative.x;
				//Find bin start and end
				unsigned int binHash = getHash(glm::ivec2(currentBinX, currentBinY));
				//if (binHash>d_gridDim*d_gridDim)
				//{
				//    printf("Hash: %d, gridDim: %d, pos: (%d, %d)\n", binHash, d_gridDim, tGridPos.x, tGridPos.y);
				//}
				unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
				unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
#else

			int currentBinX = gridPos.x - 1;
			currentBinX = currentBinX >= 0 ? currentBinX : 0;
			unsigned int binHash = getHash(glm::ivec2(currentBinX, currentBinY));
			unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
			currentBinX = gridPos.x + 1;
			currentBinX = currentBinX < d_gridDim ? currentBinX : d_gridDim-1;
			binHash = getHash(glm::ivec2(currentBinX, currentBinY));
			unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
#endif
			//Iterate messages in range
			for (unsigned int i = binStart; i < binEnd; ++i)
			{
				//if (i != index)//Ignore self
				{
					float2 message = tex1Dfetch(d_texMessages, i); 
					//if (gridPos.x == 3 && gridPos.y == 3)
					//	printf("%d\n", index);
					//if (gridPos.x == 3 && gridPos.y == 3 && index == 1058)
					//if (index == 9)
					//	printf("(%.3f, %.3f)\n", message.x, message.y);
#ifndef CIRCLES
					if (distance(*(glm::vec2*)&message, pos) < d_RADIUS)
					{
						//message.z = pow(sqrt(sin(distance(message, pos))),3.1f);//Bonus compute
						average += *(glm::vec2*)&message;
						count++;
					}
#else
					glm::vec2 toLoc = (*(glm::vec2*)&message) - pos;//Difference
					float separation = length(toLoc);
					if (separation < d_RADIUS && separation > 0)
					{
						const float REPULSE_FACTOR = 0.05f;
						float k = sinf((separation / d_RADIUS)*3.141*-2)*REPULSE_FACTOR;
						toLoc /= separation;//Normalize (without recalculating seperation)
						average += k * toLoc;
						count++;
					}
#endif
				}
			}
		}
#ifndef STRIPS
	}
#endif
}
average /= count>0 ? count : 1;
#ifndef CIRCLES
out[index] = average;
#else
out[index] = pos + average;
#endif
}
/**
 * Kernel must be launched 1 block per bin
 * This removes the necessity of __launch_bounds__(64) as all threads in block are touching the same messages
 * However we end up with alot of (mostly) idle threads if one bin dense, others empty.
 */
__global__ void neighbourSearch(const glm::vec2 *agents, glm::vec2 *out)
{
	extern __shared__ float2 sm_messages[];
	unsigned int *stripStarts = (unsigned int *)&sm_messages[d_SHARED_MESSAGE_COUNT];
	unsigned int *stripCounts = &stripStarts[3];


	//My data
	glm::ivec2 myBin = glm::ivec2(blockIdx.x, blockIdx.y);
	unsigned int index = UINT_MAX;
	glm::vec2 pos;
	{
		unsigned int binHash = getHash(myBin);
		unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
		unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
		unsigned int binCount = binEnd - binStart;
		if (threadIdx.x < binCount)
		{
			index = binStart + threadIdx.x;
			pos = agents[index];
		}
	}
	//PBM data?
	//How do we decide which threads load which messages?
	//We can do 6 accesses to PBM (in 2D), to identify the 3 Strips
	if(threadIdx.x<3)
	{
		int myY = myBin.y + ((int)threadIdx.x) - 1;
		if (myY >= 0 && myY < d_gridDim)
		{
			int currentBinX = myBin.x - 1;
			currentBinX = currentBinX >= 0 ? currentBinX : 0;
			unsigned int binHash = getHash(glm::ivec2(currentBinX, myY));
			unsigned int binStart = tex1Dfetch(d_texPBM, binHash);
			stripStarts[threadIdx.x] = binStart;
			currentBinX = myBin.x + 1;
			currentBinX = currentBinX < d_gridDim ? currentBinX : d_gridDim - 1;
			binHash = getHash(glm::ivec2(currentBinX, myY));
			unsigned int binEnd = tex1Dfetch(d_texPBM, binHash + 1);
			stripCounts[threadIdx.x] = binEnd - binStart;
		}
		else
			stripCounts[threadIdx.x] = 0;
	}
	__syncthreads();

	const unsigned int TOTAL_LOAD_COUNT = stripCounts[0] + stripCounts[1] + stripCounts[2];

	//if (blockIdx.x == 2 && blockIdx.y == 0 && threadIdx.x == 0)
	//	printf("Total loads due: %d [%d, %d, %d]\n", TOTAL_LOAD_COUNT, stripCounts[0], stripCounts[1], stripCounts[2]);
	{
		//Model data
		unsigned int count = 0;
		glm::vec2 average = glm::vec2(0);
		//Loading data
		unsigned int myStrip = 0;
		int myLoad = threadIdx.x;
		//TOTAL_LOAD_COUNT: Identifies the total number of messages in the Moore neighbourhood
		//blockLoad: Identifies the index of the first thread through the entire Moore neighbourhood of messages to be accessed
		for(unsigned int blockLoad = 0; blockLoad<TOTAL_LOAD_COUNT; blockLoad += d_SHARED_MESSAGE_COUNT)
		{
			//Load the corresponding message if available
			if(threadIdx.x<d_SHARED_MESSAGE_COUNT)
			{
				//Get us pointing to a valid index in a valid strip
				unsigned int stripCount = stripCounts[myStrip];
				//If we exceed the index of current strip
				while(myLoad>=stripCount&&myStrip<3)
				{
					//Reduce our index by stripCount
					myLoad -= stripCount;
					//Switch to next strip
					myStrip++;
					//updateStripCount
					if(myStrip<3)
						stripCount = stripCounts[myStrip];
				}
				//If we're still valid
				if (myStrip < 3)
				{
					//Load Message
					sm_messages[threadIdx.x] = tex1Dfetch(d_texMessages, stripStarts[myStrip] + myLoad);

					//if (blockIdx.x == 3 && blockIdx.y == 3)
					//	printf("(%.3f, %.3f) = %d:%d\n", sm_messages[threadIdx.x].x, sm_messages[threadIdx.x].y, myStrip, myLoad);
					//Prep for next loop
					myLoad += d_SHARED_MESSAGE_COUNT;// blockDim.x;
				}
			}
			//Wait for shared mem to be filled
			__syncthreads();
			//Loop all loaded messages
			if(index != UINT_MAX)
			{
				unsigned int blockLeft = min(TOTAL_LOAD_COUNT - blockLoad, d_SHARED_MESSAGE_COUNT);//Either loop all messages, or remaining messages
				for (unsigned int i = 0; i<blockLeft; ++i)
				{
					float2 message = sm_messages[i];
					//if (blockIdx.x == 2 && blockIdx.y == 0 && threadIdx.x == 0)
					//	printf("(%.3f, %.3f)\n", message.x, message.y);
#ifndef CIRCLES
					if (distance(*(glm::vec2*)&message, pos)<d_RADIUS)
					{
						//message.z = pow(sqrt(sin(distance(message, pos))),3.1f);//Bonus compute
						average += *(glm::vec2*)&message;
						count++;
					}
#else
					glm::vec2 toLoc = (*(glm::vec2*)&message) - pos;//Difference
					float separation = length(toLoc);
					if (separation < d_RADIUS && separation > 0)
					{
						const float REPULSE_FACTOR = 0.05f;
						float k = sinf((separation / d_RADIUS)*3.141*-2)*REPULSE_FACTOR;
						toLoc /= separation;//Normalize (without recalculating seperation)
						average += k * toLoc;
						count++;
					}
#endif
				}
			}
			//Wait for threads to finish using shared mem
			if(blockLoad + d_SHARED_MESSAGE_COUNT<TOTAL_LOAD_COUNT)
			{
				__syncthreads();
			}
		}
		//If we have a valid message...
		if (index != UINT_MAX)
		{
#ifndef CIRCLES
			average /= count>0 ? count : 1;
			out[index] = average;
#else
			out[index] = pos + average;
#endif
		}
	}

}


__global__ void unsortMessages(
	unsigned int* bin_index,
	unsigned int* bin_sub_index,
	unsigned int *pbm,
	glm::vec2 *ordered_messages,
	glm::vec2 *unordered_messages
)
{
	unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//Kill excess threads
	if (index >= d_agentCount) return;

	unsigned int i = bin_index[index];
	unsigned int sorted_index = pbm[i] + bin_sub_index[index];

	//Order messages into swap space
	unordered_messages[index] = ordered_messages[sorted_index];
}
/**
* This program is to act as a test rig to demonstrate the raw impact of raw message handling
*/
void run(std::ofstream &f, const unsigned int ENV_WIDTH, const unsigned int AGENT_COUNT = 1000000)
{
	void *d_CUB_temp_storage = nullptr;
	size_t d_CUB_temp_storage_bytes = 0;
	//Spatial partitioning mock
	//Fixed 2D environment of 1000x1000
	//Filled with 1,000,000 randomly distributed agents
	//const unsigned int ENV_WIDTH = 250;
	float ENV_WIDTH_float = (float)ENV_WIDTH;
	const unsigned int RNG_SEED = 12;
	const unsigned int ENV_VOLUME = ENV_WIDTH * ENV_WIDTH;
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_agentCount), &AGENT_COUNT, sizeof(unsigned int)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_environmentWidth_float), &ENV_WIDTH_float, sizeof(float)));
	glm::vec2 *d_agents_init = nullptr, *d_agents = nullptr, *d_out = nullptr;
	unsigned int *d_keys = nullptr, *d_vals = nullptr;
	CUDA_CALL(hipMalloc(&d_agents_init, sizeof(glm::vec2) * AGENT_COUNT));
	CUDA_CALL(hipMalloc(&d_agents, sizeof(glm::vec2) * AGENT_COUNT));
	CUDA_CALL(hipMalloc(&d_out, sizeof(glm::vec2) * AGENT_COUNT));
	glm::vec2 *h_out = (glm::vec2*)malloc(sizeof(glm::vec2) * AGENT_COUNT);
	glm::vec2 *h_out_control = (glm::vec2*)malloc(sizeof(glm::vec2) * AGENT_COUNT);
	//Init agents
	{
		//Generate hiprand
		hiprandState *d_rng;
		CUDA_CALL(hipMalloc(&d_rng, AGENT_COUNT * sizeof(hiprandState)));
		//Arbitrary thread block sizes (speed not too important during one off initialisation)
		unsigned int initThreads = 512;
		unsigned int initBlocks = (AGENT_COUNT / initThreads) + 1;
		init_curand << <initBlocks, initThreads >> >(d_rng, RNG_SEED);//Defined in CircleKernels.cuh
		CUDA_CALL(hipDeviceSynchronize());
		init_agents << <initBlocks, initThreads >> >(d_rng, d_agents_init);
		//Free hiprand
		CUDA_CALL(hipFree(d_rng));
		CUDA_CALL(hipMalloc(&d_keys, sizeof(unsigned int)*AGENT_COUNT));
		CUDA_CALL(hipMalloc(&d_vals, sizeof(unsigned int)*AGENT_COUNT));
	}
	//Decide interaction radius
	//for a range of bin widths
	const float RADIUS = 1.0f;//
	const float RADIAL_VOLUME = glm::pi<float>()*RADIUS*RADIUS;
	const unsigned int AVERAGE_NEIGHBOURS = (unsigned int)(AGENT_COUNT*RADIAL_VOLUME / ENV_VOLUME);
	printf("Agents: %d, RVol: %.2f, Average Neighbours: %d\n", AGENT_COUNT, RADIAL_VOLUME, AVERAGE_NEIGHBOURS);
	//{
	//    hipFree(d_agents_init);
	//    hipFree(d_agents);
	//    hipFree(d_out);
	//    return;
	//}
	//Decide how many messages we can fit into shared memory at once
	{
		hipDeviceProp_t dp;
		int device;
		hipGetDevice(&device);
		memset(&dp, sizeof(hipDeviceProp_t), 0);
		hipGetDeviceProperties(&dp, device);
		//We could use dp.sharedMemPerBlock/N to improve occupancy
		SHARED_MESSAGE_COUNT = (dp.sharedMemPerBlock-(sizeof(unsigned int)*6)) / sizeof(float2);
		SHARED_MESSAGE_COUNT = glm::min(SHARED_MESSAGE_COUNT, 256u);
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_SHARED_MESSAGE_COUNT), &SHARED_MESSAGE_COUNT, sizeof(unsigned int)));
	}

	const float rSin45 = (float)(RADIUS*sin(glm::radians(45)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_RADIUS), &RADIUS, sizeof(float)));
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_R_SIN_45), &rSin45, sizeof(float)));
	{
		{
			//Copy init state to d_out   
			CUDA_CALL(hipMemcpy(d_out, d_agents_init, sizeof(glm::vec2)*AGENT_COUNT, hipMemcpyDeviceToDevice));
		}
		//Decide bin width (as a ratio to radius)
		const float BIN_WIDTH = RADIUS;
		float GRID_DIMS_float = ENV_WIDTH / BIN_WIDTH;
		GRID_DIMS = glm::uvec2((unsigned int)ceil(GRID_DIMS_float));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_binWidth), &BIN_WIDTH, sizeof(float)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim), &GRID_DIMS.x, sizeof(unsigned int)));
		CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_gridDim_float), &GRID_DIMS_float, sizeof(float)));
		const unsigned int BIN_COUNT = glm::compMul(GRID_DIMS);
		hipEvent_t start_PBM, end_PBM, start_kernel, end_kernel;
		hipEventCreate(&start_PBM);
		hipEventCreate(&end_PBM);
		hipEventCreate(&start_kernel);
		hipEventCreate(&end_kernel);
		//BuildPBM
		unsigned int *d_PBM_counts = nullptr;
		unsigned int *d_PBM = nullptr;
		CUDA_CALL(hipMalloc(&d_PBM_counts, (BIN_COUNT + 1) * sizeof(unsigned int)));
		CUDA_CALL(hipMalloc(&d_PBM, (BIN_COUNT + 1) * sizeof(unsigned int)));
		//Prep for threadblocks
		CUDA_CALL(hipMalloc(&d_PBM_max_count, sizeof(unsigned int)));
		CUDA_CALL(hipMemset(d_PBM_max_count, 0, sizeof(unsigned int)));
		{//Resize cub temp if required
			size_t bytesCheck, bytesCheck2;
			hipcub::DeviceScan::ExclusiveSum(nullptr, bytesCheck, d_PBM, d_PBM_counts, BIN_COUNT + 1);
			hipcub::DeviceReduce::Max(nullptr, bytesCheck2, d_PBM_counts, d_PBM_max_count, BIN_COUNT);
			bytesCheck = glm::max(bytesCheck, bytesCheck2);
			if (bytesCheck > d_CUB_temp_storage_bytes)
			{
				if (d_CUB_temp_storage)
				{
					CUDA_CALL(hipFree(d_CUB_temp_storage));
				}
				d_CUB_temp_storage_bytes = bytesCheck;
				CUDA_CALL(hipMalloc(&d_CUB_temp_storage, d_CUB_temp_storage_bytes));
			}
		}

		float pbmMillis_control = 0, kernelMillis_control = 0;
		float pbmMillis = 0, kernelMillis = 0;
		for (unsigned int _j = 1; _j < UINT_MAX; --_j)
		{
			//1 = control
			//0 = threadblock
			bool isControl = _j != 0;

			//For 200 iterations (to produce an average)
			const unsigned int ITERATIONS = 1;
			for (unsigned int i = 0; i < ITERATIONS; ++i)
			{
				//Reset each run of average model
#ifndef CIRCLES
				CUDA_CALL(hipMemcpy(d_out, d_agents_init, sizeof(glm::vec2)*AGENT_COUNT, hipMemcpyDeviceToDevice));
#endif	
				hipEventRecord(start_PBM);
				{//Build atomic histogram
					CUDA_CALL(hipMemset(d_PBM_counts, 0x00000000, (BIN_COUNT + 1) * sizeof(unsigned int)));
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, atomicHistogram, 32, 0));//Randomly 32
																												 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					atomicHistogram << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM_counts, d_out);
					CUDA_CALL(hipDeviceSynchronize());
				}
				{//Scan (sum), to finalise PBM
					hipcub::DeviceScan::ExclusiveSum(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_PBM_counts, d_PBM, BIN_COUNT + 1);
				}
				{//Reorder messages
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
																														 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					//Copy messages from d_messages to d_messages_swap, in hash order
					reorderLocationMessages << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM, d_out, d_agents);
					CUDA_CHECK();
				}
				if (!isControl)
				{//Calc max bin size (for threadblocks)
					hipcub::DeviceReduce::Max(d_CUB_temp_storage, d_CUB_temp_storage_bytes, d_PBM_counts, d_PBM_max_count, BIN_COUNT);
					CUDA_CALL(hipGetLastError());
					CUDA_CALL(hipMemcpy(&PBM_max_count, d_PBM_max_count, sizeof(unsigned int), hipMemcpyDeviceToHost));
					//Calc moore size (bin size^dims?)
					//PBM_max_Moore_count = (unsigned int)pow(PBM_max_count, 2);//2==2D//Unused, requires 9x shared mem in 2D, 27x in 3D
				}
				{//Fill PBM and Message Texture Buffers																			  
					CUDA_CALL(hipDeviceSynchronize());//Wait for return
					CUDA_CALL(hipBindTexture(nullptr, d_texMessages, d_agents, sizeof(glm::vec2) * AGENT_COUNT));
					CUDA_CALL(hipBindTexture(nullptr, d_texPBM, d_PBM, sizeof(unsigned int) * (BIN_COUNT + 1)));
				}
				hipEventRecord(end_PBM);
				hipEventRecord(start_kernel);
				if (isControl)
				{
					//Each message samples radial neighbours (static model)
					int blockSize;   // The launch configurator returned block size 
					CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
					 // Round up according to array size
					int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
					//Copy messages from d_agents to d_out, in hash order
					printf("Control:\n");
					neighbourSearch_control << <gridSize, blockSize >> > (d_agents, d_out);
					CUDA_CHECK();
				}
				else
				{
					//Each message samples radial neighbours (static model)
					int blockSize = glm::max(PBM_max_count, SHARED_MESSAGE_COUNT);   //blockSize == largest bin size
					dim3 gridSize;
					gridSize.x = GRID_DIMS.x;
					gridSize.y = GRID_DIMS.y;
					gridSize.z = 1;// GRID_DIMS.z;
					//Copy messages from d_agents to d_out, in hash order
					printf("Test:\n");
					neighbourSearch << <gridSize, blockSize, requiredSM(blockSize) >> > (d_agents, d_out);
					CUDA_CHECK();
				}
				CUDA_CALL(hipDeviceSynchronize());
				hipEventRecord(end_kernel);
				hipEventSynchronize(end_kernel);

				float _pbmMillis = 0, _kernelMillis = 0;
				hipEventElapsedTime(&_pbmMillis, start_PBM, end_PBM);
				hipEventElapsedTime(&_kernelMillis, start_kernel, end_kernel);
				if (isControl)
				{
					pbmMillis_control += _pbmMillis;
					kernelMillis_control += _kernelMillis;
				}
				else
				{
					pbmMillis += _pbmMillis;
					kernelMillis += _kernelMillis;
				}

			}//for(ITERATIONS)
			pbmMillis_control /= ITERATIONS;
			kernelMillis_control /= ITERATIONS;
			pbmMillis /= ITERATIONS;
			kernelMillis /= ITERATIONS;

			{//Unorder messages
				int blockSize;   // The launch configurator returned block size 
				CUDA_CALL(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blockSize, reorderLocationMessages, 32, 0));//Randomly 32
																													 // Round up according to array size
				int gridSize = (AGENT_COUNT + blockSize - 1) / blockSize;
				//Copy messages from d_out to d_agents, in hash order
				unsortMessages << <gridSize, blockSize >> > (d_keys, d_vals, d_PBM, d_out, d_agents);
				CUDA_CHECK();
				//Swap d_out and d_agents
				{
					glm::vec2 *t = d_out;
					d_out = d_agents;
					d_agents = t;
				}
				//Wait for return
				CUDA_CALL(hipDeviceSynchronize());
				//Copy back to relative host array (for validation)
				CUDA_CALL(hipMemcpy(isControl ? h_out_control : h_out, d_out, sizeof(glm::vec2)*AGENT_COUNT, hipMemcpyDeviceToHost));
				CUDA_CALL(hipDeviceSynchronize());
			}
		}//for(MODE)
		CUDA_CALL(hipUnbindTexture(d_texPBM));
		CUDA_CALL(hipUnbindTexture(d_texMessages));
		CUDA_CALL(hipFree(d_PBM_counts));
		CUDA_CALL(hipFree(d_PBM));
		//log();
		printf("Control:     PBM: %.2fms, Kernel: %.2fms\n", pbmMillis_control, kernelMillis_control);
		printf("ThreadBlock: PBM: %.2fms, Kernel: %.2fms\n", pbmMillis, kernelMillis);
		unsigned int fails = 0;
#ifndef CIRCLES

		{//Validation
			//Validate results for average model
			//thrust::sort(thrust::cuda::par, d_out, d_out + AGENT_COUNT, vec2Compare());
			//CUDA_CALL(hipMemcpy(isControl ? h_out_control : h_out, d_out, sizeof(glm::vec2)*AGENT_COUNT, hipMemcpyDeviceToHost));
			for (unsigned int i = 0; i < AGENT_COUNT; ++i)
			{
				assert(!(isnan(h_out[i].x) || isnan(h_out[i].y)));
				if (isnan(h_out[i].x) || isnan(h_out[i].y))
					printf("err nan\n");
				auto ret = glm::epsilonEqual(h_out[i], h_out_control[i], EPSILON);
				if (!(ret.x&&ret.y))
				{
					if (fails == 0)
						printf("#%d: (%.5f, %.5f) vs (%.5f, %.5f)\n", i, h_out_control[i].x, h_out_control[i].y, h_out[i].x, h_out[i].y);
					fails++;
				}
			}
			if (fails > 0)
				printf("%d/%d (%.1f%%) Failed.\n", fails, AGENT_COUNT, 100 * (fails / (float)AGENT_COUNT));
		}
#endif
		log(f, AVERAGE_NEIGHBOURS, AGENT_COUNT, ENV_WIDTH, pbmMillis_control, kernelMillis_control, pbmMillis, kernelMillis, fails);
	}

	CUDA_CALL(hipUnbindTexture(d_texMessages));
	CUDA_CALL(hipFree(d_vals));
	CUDA_CALL(hipFree(d_keys));
	CUDA_CALL(hipFree(d_agents));
	CUDA_CALL(hipFree(d_agents_init));
	CUDA_CALL(hipFree(d_out));
	free(h_out);
	free(h_out_control);
}
void runAgents(std::ofstream &f, const unsigned int AGENT_COUNT, const float DENSITY)
{
	//density refers to approximate number of neighbours
	run(f, (unsigned int)sqrt(AGENT_COUNT / (DENSITY*2.86 / 9)), AGENT_COUNT);
}
int main()
{
	{
		std::ofstream f;
		createLog(f);
		assert(f.is_open());
		for (unsigned int i = 20000; i <= 3000000; i += 20000)
		{
			//Run i agents in a density with roughly 60 radial neighbours, and log
			//Within this, it is tested over a range of proportional bin widths
			runAgents(f, i, 20);
			break;
		}
	}
	printf("fin\n");
	getchar();
	return 0;
}

